#include "hip/hip_runtime.h"
/**
    The purpose of this algorithm is not to design a clean CUDA-code which performs
    checks on input data and cuda error recovery.
    
    The idea is just to design a simple way of dealing with CUDA and non-CUDA computers using a single sourcecode
    
    __HIPCC__: defines whether nvcc is steering compilation or not
    __CUDA_ARCH__: is always undefined when compiling host code, steered by nvcc or not
    __CUDA_ARCH__: is only defined for the device code trajectory of compilation steered by nvcc
    //for further details: http://stackoverflow.com/questions/8796369/cuda-and-nvcc-using-the-preprocessor-to-choose-between-float-or-double
*/

#include "cudanoncuda-src.h"
#include <stdio.h>

void init_numpy();
PyArrayObject *c_add(const double* v1, const double* v2, const unsigned int &size);
#ifdef __HIPCC__
    void c_add_cuda(const double* h_v1, const double* h_v2, const unsigned int &size, double* h_vres);
    __global__ void c_add_cuda_kernel(const double* d_v1, const double* d_v2, const unsigned int size, double* d_vres);
#else
    void c_add_cpu(const double* v1, const double* v2, const unsigned int &size, double* vres);
#endif

/**
    import_array has to be call
    before the first call to NumPy API
*/
int is_init(0);
void init_numpy()
{
    if (! is_init)
    {
        import_array();
        is_init = 1;
    }
}

PyArrayObject *c_add(const double* v1, const double* v2, const unsigned int &size)
{
    printf("PyArrayObject *c_add(double*, double*, const unsigned int&)\n");
    init_numpy(); // init NumPy if not already done
    
    // Define NumPy array
    int dims[] = {size};
    PyArrayObject *vres = (PyArrayObject *) PyArray_FromDims(1, dims, NPY_DOUBLE);
    
    #ifdef __HIPCC__
        // Apply CUDA version of c_add
        c_add_cuda(v1, v2, size, (double*)vres->data);
    #else
        // Apply GPU version of c_add
        c_add_cpu(v1, v2, size, (double*)vres->data);
    #endif
    
    return vres;
}

#ifdef __HIPCC__

void c_add_cuda(const double* h_v1, const double* h_v2, const unsigned int &size, double* h_vres)
{
    // Build CUDA copies of the host arrays
    double *d_v1, *d_v2, *d_vres;
    hipMalloc(&d_v1, size * sizeof(double)); // malloc + memcpy
    hipMemcpy(d_v1, h_v1, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&d_v2, size * sizeof(double)); // malloc + memcpy
    hipMemcpy(d_v2, h_v2, size * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&d_vres, size * sizeof(double)); // only malloc
    
    // Run CUDA kernel
    c_add_cuda_kernel<<<(size + MAX_THREADS -1)/MAX_THREADS, MAX_THREADS>>>(d_v1, d_v2, size, d_vres);
    hipDeviceSynchronize(); // block until the device is finished
    
    // Copy result to CPU
    hipMemcpy(h_vres, d_vres, size * sizeof(double), hipMemcpyDeviceToHost);
    
    // Free CUDA copies
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_vres);
}

__global__ void c_add_cuda_kernel(const double* d_v1, const double* d_v2, const unsigned int size, double* d_vres)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    d_vres[i] = d_v1[i] + d_v2[i];    
}

#else

void c_add_cpu(const double* v1, const double* v2, const unsigned int &size, double* vres)
{
    for (unsigned int i(0) ; i != size ; i++)
        vres[i] = v1[i] + v2[i];
}

#endif

